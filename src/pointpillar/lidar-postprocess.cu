#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: MIT
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "lidar-postprocess.hpp"
#include <algorithm>
#include <math.h>

namespace pointpillar {
namespace lidar {

__device__ float sigmoid(const float x) { return 1.0f / (1.0f + expf(-x)); }

__global__ void postprocess_kernal(const float *cls_input,
                                        float *box_input,
                                        const float *dir_cls_input,
                                        float *anchors,
                                        float *anchor_bottom_heights,
                                        float *BoundingBox_output,
                                        int *object_counter,
                                        const float min_x_range,
                                        const float max_x_range,
                                        const float min_y_range,
                                        const float max_y_range,
                                        const int feature_x_size,
                                        const int feature_y_size,
                                        const int num_anchors,
                                        const int num_classes,
                                        const int num_box_values,
                                        const float score_thresh,
                                        const float dir_offset)
{
  int loc_index = blockIdx.x;
  int ith_anchor = threadIdx.x;
  if (ith_anchor >= num_anchors)
  {
      return;
  }
  int col = loc_index % feature_x_size;
  int row = loc_index / feature_x_size;
  float x_offset = min_x_range + col * (max_x_range - min_x_range) / (feature_x_size - 1);
  float y_offset = min_y_range + row * (max_y_range - min_y_range) / (feature_y_size - 1);
  int cls_offset = loc_index * num_anchors * num_classes + ith_anchor * num_classes;
  float dev_cls[2] = {-1, 0};

  const float *scores = cls_input + cls_offset;
  float max_score = sigmoid(scores[0]);
  int cls_id = 0;
  for (int i = 1; i < num_classes; i++) {
    float cls_score = sigmoid(scores[i]);
    if (cls_score > max_score) {
      max_score = cls_score;
      cls_id = i;
    }
  }
  dev_cls[0] = static_cast<float>(cls_id);
  dev_cls[1] = max_score;

  if (dev_cls[1] >= score_thresh)
  {
    int box_offset = loc_index * num_anchors * num_box_values + ith_anchor * num_box_values;
    int dir_cls_offset = loc_index * num_anchors * 2 + ith_anchor * 2;
    float *anchor_ptr = anchors + ith_anchor * 4;
    float z_offset = anchor_ptr[2] / 2 + anchor_bottom_heights[ith_anchor / 2];
    float anchor[7] = {x_offset, y_offset, z_offset, anchor_ptr[0], anchor_ptr[1], anchor_ptr[2], anchor_ptr[3]};
    float *box_encodings = box_input + box_offset;

    float xa = anchor[0];
    float ya = anchor[1];
    float za = anchor[2];
    float dxa = anchor[3];
    float dya = anchor[4];
    float dza = anchor[5];
    float ra = anchor[6];
    float diagonal = sqrtf(dxa * dxa + dya * dya);
    box_encodings[0] = box_encodings[0] * diagonal + xa;
    box_encodings[1] = box_encodings[1] * diagonal + ya;
    box_encodings[2] = box_encodings[2] * dza + za;
    box_encodings[3] = expf(box_encodings[3]) * dxa;
    box_encodings[4] = expf(box_encodings[4]) * dya;
    box_encodings[5] = expf(box_encodings[5]) * dza;
    box_encodings[6] = box_encodings[6] + ra;

    float yaw;
    int dir_label = dir_cls_input[dir_cls_offset] > dir_cls_input[dir_cls_offset + 1] ? 0 : 1;
    float period = 2 * M_PI / 2;
    float val = box_input[box_offset + 6] - dir_offset;
    float dir_rot = val - floor(val / (period + 1e-8) + 0.f) * period;
    yaw = dir_rot + dir_offset + period * dir_label;

    int resCount = (int)atomicAdd(object_counter, 1);
    BoundingBox_output[0] = resCount+1;
    float *data = BoundingBox_output + 1 + resCount * 9;
    data[0] = box_input[box_offset];
    data[1] = box_input[box_offset + 1];
    data[2] = box_input[box_offset + 2];
    data[3] = box_input[box_offset + 3];
    data[4] = box_input[box_offset + 4];
    data[5] = box_input[box_offset + 5];
    data[6] = yaw;
    data[7] = dev_cls[0];
    data[8] = dev_cls[1];
  }
}

hipError_t postprocess_launch(const float *cls_input,
                      float *box_input,
                      const float *dir_cls_input,
                      float *anchors,
                      float *anchor_bottom_heights,
                      float *BoundingBox_output,
                      int *object_counter,
                      const float min_x_range,
                      const float max_x_range,
                      const float min_y_range,
                      const float max_y_range,
                      const int feature_x_size,
                      const int feature_y_size,
                      const int num_anchors,
                      const int num_classes,
                      const int num_box_values,
                      const float score_thresh,
                      const float dir_offset,
                      hipStream_t stream)
{
  int bev_size = feature_x_size * feature_y_size;
  dim3 threads (num_anchors);
  dim3 blocks (bev_size);

  postprocess_kernal<<<blocks, threads, 0, stream>>>
                (cls_input,
                 box_input,
                 dir_cls_input,
                 anchors,
                 anchor_bottom_heights,
                 BoundingBox_output,
                 object_counter,
                 min_x_range,
                 max_x_range,
                 min_y_range,
                 max_y_range,
                 feature_x_size,
                 feature_y_size,
                 num_anchors,
                 num_classes,
                 num_box_values,
                 score_thresh,
                 dir_offset);
  return hipGetLastError();
}

const float ThresHold = 1e-8;

inline float cross(const float2 p1, const float2 p2, const float2 p0) {
    return (p1.x - p0.x) * (p2.y - p0.y) - (p2.x - p0.x) * (p1.y - p0.y);
}

inline int check_box2d(const BoundingBox box, const float2 p) {
    const float MARGIN = 1e-2;
    float center_x = box.x;
    float center_y = box.y;
    float angle_cos = cos(-box.rt);
    float angle_sin = sin(-box.rt);
    float rot_x = (p.x - center_x) * angle_cos + (p.y - center_y) * (-angle_sin);
    float rot_y = (p.x - center_x) * angle_sin + (p.y - center_y) * angle_cos;

    return (fabs(rot_x) < box.w / 2 + MARGIN && fabs(rot_y) < box.l / 2 + MARGIN);
}

bool intersection(const float2 p1, const float2 p0, const float2 q1, const float2 q0, float2 &ans) {

    if (( std::min(p0.x, p1.x) <= std::max(q0.x, q1.x) &&
          std::min(q0.x, q1.x) <= std::max(p0.x, p1.x) &&
          std::min(p0.y, p1.y) <= std::max(q0.y, q1.y) &&
          std::min(q0.y, q1.y) <= std::max(p0.y, p1.y) ) == 0)
        return false;


    float s1 = cross(q0, p1, p0);
    float s2 = cross(p1, q1, p0);
    float s3 = cross(p0, q1, q0);
    float s4 = cross(q1, p1, q0);

    if (!(s1 * s2 > 0 && s3 * s4 > 0))
        return false;

    float s5 = cross(q1, p1, p0);
    if (fabs(s5 - s1) > ThresHold) {
        ans.x = (s5 * q0.x - s1 * q1.x) / (s5 - s1);
        ans.y = (s5 * q0.y - s1 * q1.y) / (s5 - s1);

    } else {
        float a0 = p0.y - p1.y, b0 = p1.x - p0.x, c0 = p0.x * p1.y - p1.x * p0.y;
        float a1 = q0.y - q1.y, b1 = q1.x - q0.x, c1 = q0.x * q1.y - q1.x * q0.y;
        float D = a0 * b1 - a1 * b0;

        ans.x = (b0 * c1 - b1 * c0) / D;
        ans.y = (a1 * c0 - a0 * c1) / D;
    }

    return true;
}

inline void rotate_around_center(const float2 &center, const float angle_cos, const float angle_sin, float2 &p) {
    float new_x = (p.x - center.x) * angle_cos + (p.y - center.y) * (-angle_sin) + center.x;
    float new_y = (p.x - center.x) * angle_sin + (p.y - center.y) * angle_cos + center.y;
    p = float2 {new_x, new_y};
    return;
}

inline float box_overlap(const BoundingBox &box_a, const BoundingBox &box_b) {
    float a_angle = box_a.rt, b_angle = box_b.rt;
    float a_dx_half = box_a.w / 2, b_dx_half = box_b.w / 2, a_dy_half = box_a.l / 2, b_dy_half = box_b.l / 2;
    float a_x1 = box_a.x - a_dx_half, a_y1 = box_a.y - a_dy_half;
    float a_x2 = box_a.x + a_dx_half, a_y2 = box_a.y + a_dy_half;
    float b_x1 = box_b.x - b_dx_half, b_y1 = box_b.y - b_dy_half;
    float b_x2 = box_b.x + b_dx_half, b_y2 = box_b.y + b_dy_half;
    float2 box_a_corners[5];
    float2 box_b_corners[5];

    float2 center_a = float2 {box_a.x, box_a.y};
    float2 center_b = float2 {box_b.x, box_b.y};

    float2 cross_points[16];
    float2 poly_center =  {0, 0};
    int cnt = 0;
    bool flag = false;

    box_a_corners[0] = float2 {a_x1, a_y1};
    box_a_corners[1] = float2 {a_x2, a_y1};
    box_a_corners[2] = float2 {a_x2, a_y2};
    box_a_corners[3] = float2 {a_x1, a_y2};

    box_b_corners[0] = float2 {b_x1, b_y1};
    box_b_corners[1] = float2 {b_x2, b_y1};
    box_b_corners[2] = float2 {b_x2, b_y2};
    box_b_corners[3] = float2 {b_x1, b_y2};

    float a_angle_cos = cos(a_angle), a_angle_sin = sin(a_angle);
    float b_angle_cos = cos(b_angle), b_angle_sin = sin(b_angle);

    for (int k = 0; k < 4; k++) {
        rotate_around_center(center_a, a_angle_cos, a_angle_sin, box_a_corners[k]);
        rotate_around_center(center_b, b_angle_cos, b_angle_sin, box_b_corners[k]);
    }

    box_a_corners[4] = box_a_corners[0];
    box_b_corners[4] = box_b_corners[0];

    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            flag = intersection(box_a_corners[i + 1], box_a_corners[i],
                                box_b_corners[j + 1], box_b_corners[j],
                                cross_points[cnt]);
            if (flag) {
                poly_center = {poly_center.x + cross_points[cnt].x, poly_center.y + cross_points[cnt].y};
                cnt++;
            }
        }
    }

    for (int k = 0; k < 4; k++) {
        if (check_box2d(box_a, box_b_corners[k])) {
            poly_center = {poly_center.x + box_b_corners[k].x, poly_center.y + box_b_corners[k].y};
            cross_points[cnt] = box_b_corners[k];
            cnt++;
        }
        if (check_box2d(box_b, box_a_corners[k])) {
            poly_center = {poly_center.x + box_a_corners[k].x, poly_center.y + box_a_corners[k].y};
            cross_points[cnt] = box_a_corners[k];
            cnt++;
        }
    }

    poly_center.x /= cnt;
    poly_center.y /= cnt;

    float2 temp;
    for (int j = 0; j < cnt - 1; j++) {
        for (int i = 0; i < cnt - j - 1; i++) {
            if (atan2(cross_points[i].y - poly_center.y, cross_points[i].x - poly_center.x) >
                atan2(cross_points[i+1].y - poly_center.y, cross_points[i+1].x - poly_center.x)
                ) {
                temp = cross_points[i];
                cross_points[i] = cross_points[i + 1];
                cross_points[i + 1] = temp;
            }
        }
    }

    float area = 0;
    for (int k = 0; k < cnt - 1; k++) {
        float2 a = {cross_points[k].x - cross_points[0].x,
                    cross_points[k].y - cross_points[0].y};
        float2 b = {cross_points[k + 1].x - cross_points[0].x,
                    cross_points[k + 1].y - cross_points[0].y};
        area += (a.x * b.y - a.y * b.x);
    }
    return fabs(area) / 2.0;
}

int nms_cpu(std::vector<BoundingBox> BoundingBoxes, const float nms_thresh, std::vector<BoundingBox> &nms_pred)
{
    std::sort(BoundingBoxes.begin(), BoundingBoxes.end(),
              [](BoundingBox boxes1, BoundingBox boxes2) { return boxes1.score > boxes2.score; });
    std::vector<int> suppressed(BoundingBoxes.size(), 0);
    for (size_t i = 0; i < BoundingBoxes.size(); i++) {
        if (suppressed[i] == 1) {
            continue;
        }
        nms_pred.emplace_back(BoundingBoxes[i]);
        for (size_t j = i + 1; j < BoundingBoxes.size(); j++) {
            if (suppressed[j] == 1) {
                continue;
            }

            float sa = BoundingBoxes[i].w * BoundingBoxes[i].l;
            float sb = BoundingBoxes[j].w * BoundingBoxes[j].l;
            float s_overlap = box_overlap(BoundingBoxes[i], BoundingBoxes[j]);
            float iou = s_overlap / fmaxf(sa + sb - s_overlap, ThresHold);

            if (iou >= nms_thresh) {
                suppressed[j] = 1;
            }
        }
    }
    return 0;
}

PostProcessCuda::PostProcessCuda(hipStream_t stream)
{
  stream_ = stream;

  checkRuntime(hipMalloc((void **)&anchors_, params_.num_anchors * params_.len_per_anchor * sizeof(float)));
  checkRuntime(hipMalloc((void **)&anchor_bottom_heights_, params_.num_classes * sizeof(float)));
  checkRuntime(hipMalloc((void **)&object_counter_, sizeof(int)));

  checkRuntime(hipMemcpyAsync(anchors_, params_.anchors,
        params_.num_anchors * params_.len_per_anchor * sizeof(float), hipMemcpyDefault, stream_));
  checkRuntime(hipMemcpyAsync(anchor_bottom_heights_, params_.anchor_bottom_heights,
                     params_.num_classes * sizeof(float), hipMemcpyDefault, stream_));
  checkRuntime(hipMemsetAsync(object_counter_, 0, sizeof(int), stream_));
  return;
}

PostProcessCuda::~PostProcessCuda()
{
  checkRuntime(hipFree(anchors_));
  checkRuntime(hipFree(anchor_bottom_heights_));
  checkRuntime(hipFree(object_counter_));
  return;
}

int PostProcessCuda::doPostprocessCuda(const float *cls_input, float *box_input, const float *dir_cls_input,
                                        float *BoundingBox_output)
{
  checkRuntime(hipMemsetAsync(object_counter_, 0, sizeof(int)));
  checkRuntime(postprocess_launch(cls_input,
                     box_input,
                     dir_cls_input,
                     anchors_,
                     anchor_bottom_heights_,
                     BoundingBox_output,
                     object_counter_,
                     params_.min_x_range,
                     params_.max_x_range,
                     params_.min_y_range,
                     params_.max_y_range,
                     params_.feature_x_size,
                     params_.feature_y_size,
                     params_.num_anchors,
                     params_.num_classes,
                     params_.num_box_values,
                     params_.score_thresh,
                     params_.dir_offset,
                     stream_
                     ));
  return 0;
}

};  // namespace lidar
};  // namespace pointpillar